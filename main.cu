#include "hip/hip_runtime.h"
%%cu
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#define KERNEL_SIZE 1
#define OFFSET 1
#define thresh_max 90
#define thresh_min 30
#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

__constant__ double gaussian_kernel[9] = {1, 2, 1, 2, 4, 2, 1, 2, 1};
__constant__ int gaussian_kernel_sum = 16;
__constant__ int8_t Gx[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
__constant__ int8_t Gy[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

__global__ void gaussian_blur(const uint8_t *input_image, int height, int width, uint8_t *output_image)
{
    // calculate the row and col of this pixel in the image
    int row = threadIdx.x + blockIdx.x * (blockDim.x);
    int col = threadIdx.y + blockIdx.y * (blockDim.y);
    
    if (col < OFFSET || col >= width - OFFSET || row < OFFSET ||row >= height - OFFSET) return;
    
    double output_intensity = 0;
    int kernel_index = 0;
    int pixel_index = col + (row * width);
    for (int krow = -OFFSET; krow <= OFFSET; krow++)
    {
        for (int kcol = -OFFSET; kcol <= OFFSET; kcol++)
        {
            
            output_intensity += input_image[pixel_index + (kcol + (krow * width))] * gaussian_kernel[kernel_index];
            kernel_index++;
        }
    }
    output_image[pixel_index] = (uint8_t)(output_intensity / gaussian_kernel_sum);
}

__global__ void gradient_magnitude_direction(const uint8_t *input_image,int height, int width,double *magnitude,uint8_t *direction) {
  
  // calculate the row and col of this pixel in the image
   int row = threadIdx.x + blockIdx.x * (blockDim.x);
   int col = threadIdx.y + blockIdx.y * (blockDim.y);
  // if pixel is outside the given offset don't do anything
  if (col < OFFSET || col >= width - OFFSET || row < OFFSET || row >= height - OFFSET) return;

  double grad_x_sum = 0.0;
  double grad_y_sum = 0.0;
  int kernel_index = 0;
  int pixel_index = col + (row * width);

  for (int krow = -OFFSET; krow <= OFFSET; krow++) {
    for (int kcol = -OFFSET; kcol <= OFFSET; kcol++) {
      grad_x_sum +=
          input_image[pixel_index + (kcol + (krow * width))] * Gx[kernel_index];
      grad_y_sum +=
          input_image[pixel_index + (kcol + (krow * width))] * Gy[kernel_index];
      kernel_index++;
    }
  }

  int pixel_direction = 0;

  if (grad_x_sum == 0.0 || grad_y_sum == 0.0) {
    magnitude[pixel_index] = 0;
  } else {
    magnitude[pixel_index] =
        ((std::sqrt((grad_x_sum * grad_x_sum) + (grad_y_sum * grad_y_sum))));
    double theta = std::atan2(grad_y_sum, grad_x_sum);
    theta = theta * (360.0 / (2.0 * M_PI));

    if ((theta <= 22.5 && theta >= -22.5) || (theta <= -157.5) ||
        (theta >= 157.5))
      pixel_direction = 1; // horizontal -
    else if ((theta > 22.5 && theta <= 67.5) ||
             (theta > -157.5 && theta <= -112.5))
      pixel_direction = 2; // north-east -> south-west/
    else if ((theta > 67.5 && theta <= 112.5) ||
             (theta >= -112.5 && theta < -67.5))
      pixel_direction = 3; // vertical |
    else if ((theta >= -67.5 && theta < -22.5) ||
             (theta > 112.5 && theta < 157.5))
      pixel_direction = 4; // north-west -> south-east \'
  }
  direction[pixel_index] = (uint8_t)pixel_direction;
}

__global__ void non_max_suppression(double *gradient_magnitude,uint8_t *gradient_direction, int height,int width, double *output_image) {
    // calculate the row and col of this pixel in the image
   int row = threadIdx.x + blockIdx.x * (blockDim.x);
   int col = threadIdx.y + blockIdx.y * (blockDim.y);
  // if pixel is outside the given offset don't do anything
  if (col < OFFSET || col >= width - OFFSET || row < OFFSET ||
      row >= height - OFFSET)
    return;
  int pixel_index = col + (row * width);

  // unconditionally suppress border pixels
  if (row == OFFSET || col == OFFSET || col == width - OFFSET - 1 ||
      row == height - OFFSET - 1) {
    output_image[pixel_index] = 0;
    return;
  }

  switch (gradient_direction[pixel_index]) {
  case 1:
    if (gradient_magnitude[pixel_index - 1] >=
            gradient_magnitude[pixel_index] ||
        gradient_magnitude[pixel_index + 1] > gradient_magnitude[pixel_index])
      output_image[pixel_index] = 0;
    break;
  case 2:
    if (gradient_magnitude[pixel_index - (width - 1)] >=
            gradient_magnitude[pixel_index] ||
        gradient_magnitude[pixel_index + (width - 1)] >
            gradient_magnitude[pixel_index])
      output_image[pixel_index] = 0;
    break;
  case 3:
    if (gradient_magnitude[pixel_index - (width)] >=
            gradient_magnitude[pixel_index] ||
        gradient_magnitude[pixel_index + (width)] >
            gradient_magnitude[pixel_index])
      output_image[pixel_index] = 0;
    break;
  case 4:
    if (gradient_magnitude[pixel_index - (width + 1)] >=
            gradient_magnitude[pixel_index] ||
        gradient_magnitude[pixel_index + (width + 1)] >
            gradient_magnitude[pixel_index])
      output_image[pixel_index] = 0;
    break;
  default:
    output_image[pixel_index] = 0;
    break;
  }
}


__global__ void thresholding(double *suppressed_image, int height, int width,
                  int high_threshold, int low_threshold, uint8_t *output_image) {

      int row = threadIdx.x + blockIdx.x * (blockDim.x) -1 ;
      int col = threadIdx.y + blockIdx.y * (blockDim.y) -1 ;             
      int pixel_index = col + (row * width);

      if (pixel_index < 0 || pixel_index >= height * width)
      return;

      if (suppressed_image[pixel_index] > high_threshold)
        output_image[pixel_index] = 255; // Strong edge
      else if (suppressed_image[pixel_index] > low_threshold)
        output_image[pixel_index] = 100; // Weak edge
      else
        output_image[pixel_index] = 0; // Not an edge
}




__global__ void hysteresis(uint8_t *input_image, int height, int width) {

      int row = threadIdx.x + blockIdx.x * (blockDim.x);
      int col = threadIdx.y + blockIdx.y * (blockDim.y);
      int pixel_index = col + (row * width);
      if (input_image[pixel_index] == 100) {
        if (input_image[pixel_index - 1] == 255 ||
            input_image[pixel_index + 1] == 255 ||
            input_image[pixel_index - width] == 255 ||
            input_image[pixel_index + width] == 255 ||
            input_image[pixel_index - width - 1] == 255 ||
            input_image[pixel_index - width + 1] == 255 ||
            input_image[pixel_index + width - 1] == 255 ||
            input_image[pixel_index + width + 1] == 255)

          input_image[pixel_index] = 255;
        else
          input_image[pixel_index] = 0;
      }
}


void canny_edge_detect(const uint8_t *input_image, int height, int width, int high_threshold, int low_threshold, uint8_t *output_image)
{

    /*hipEvent_t start_time, stop_time;
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);
    hipEventRecord(start_time, 0);*/

    int image_size = height * width;
    dim3 block(12,12);
    dim3 grid((height+block.y-1) / block.y , (width+block.x-1) / block.x);
 
    /******** Gaussian Blur  *******************/
 
    uint8_t *gaussian_blur_input;
    uint8_t *gaussian_blur_output;
    hipMalloc((void **)&gaussian_blur_input, image_size );
    hipMalloc((void **)&gaussian_blur_output, image_size );
    hipMemcpy(gaussian_blur_input, input_image, image_size, hipMemcpyHostToDevice);
    /**"Launching gaussian blur kernel..." **/
    gaussian_blur<<<grid,block>>>(gaussian_blur_input, height, width, gaussian_blur_output);
    hipPeekAtLastError();
    hipDeviceSynchronize();
    // "Gaussian blur kernel finished."
    hipFree(gaussian_blur_input);
  /******** Gaussian Blur Done*******************/

    /********  Gradient Magnitude and Direction  *****************/
    double *gradient_magnitude;
    uint8_t *gradient_direction;

    double *gradient_magnitude_output;
    uint8_t *gradient_direction_output;

    hipMalloc((void **)&gradient_magnitude, image_size * sizeof(double*) );
    hipMalloc((void **)&gradient_direction, image_size * sizeof(double*) );
    gradient_magnitude_direction<<<grid, block>>>(gaussian_blur_output, height, width, gradient_magnitude, gradient_direction);
    hipPeekAtLastError();
    hipDeviceSynchronize();
    hipFree(gaussian_blur_output);
  /********  Gradient Magnitude and Direction  Done*****************/

  /********  Non-max Suppression        *****************/
    double *nms_output;

    hipMalloc((void **)&nms_output, image_size * sizeof(double*) );
    hipMemcpy(nms_output, gradient_magnitude,image_size * sizeof(double*) ,hipMemcpyDeviceToDevice);
    non_max_suppression<<<grid,block>>>( gradient_magnitude, gradient_direction, height, width, nms_output);
    hipPeekAtLastError();
    hipDeviceSynchronize();
    //"Non-max suppression kernel finished." << endl;
    hipFree(gradient_magnitude);
    hipFree(gradient_direction);
  /********  Non-max Suppression DONE        *****************/
 

    //thresholding
    uint8_t *threshold_output;
    hipMalloc((void **)&threshold_output, image_size );
    thresholding<<<grid,block>>>(nms_output, height, width, high_threshold, low_threshold, threshold_output);
    hipPeekAtLastError();
    hipDeviceSynchronize();
    hipFree(nms_output);

    //hysterisis 
    uint8_t *hysteresis_output;
    hipMalloc((void **)&hysteresis_output, image_size );
    hysteresis<<<grid,block>>>(threshold_output, height, width);
    hipPeekAtLastError();
    hipDeviceSynchronize();
    hipMemcpy(output_image, threshold_output, image_size, hipMemcpyDeviceToHost);
    hipFree(threshold_output);

    /*hipEventRecord(stop_time, 0);
    hipEventSynchronize(stop_time);
    float delta = 0;
    hipEventElapsedTime(&delta, start_time, stop_time);
    hipEventDestroy(start_time);
    hipEventDestroy(stop_time);
    printf("%lf\n", delta);*/
    

}

int main(int argc, char **argv)
{
      int width, height, channels, gray_channels;

      unsigned char *img = stbi_load("sky2.jpeg", &width, &height, &channels, 0);

      if (channels == 4)
          gray_channels = 2;
      else
          gray_channels = 1;

      int img_size = width * height * channels;
      int img_size_gray = width * height * gray_channels;

      unsigned char *gray_img = (unsigned char*) malloc(img_size_gray);

      if (img == NULL)
      {
          printf("Error in loading the image\n");
          exit(1);
      }
      printf("Loaded image with a width of %dpx, a height of %dpx and %d channels\n", width, height, channels);
      if (gray_img == NULL)
      {
          printf("Error in loading the gray image\n");
          exit(1);
      }

      for (int i = 0, j = 0; i != img_size; i += channels, j += gray_channels)
      {
          gray_img[j] = (uint8_t)((img[i + 0] + img[i + 1] + img[i + 2]) / 3.0);
          if (channels == 4)
              gray_img[j + 1] = img[i + 3];
      }

      stbi_write_jpg("sky_gray.jpeg", width, height, gray_channels, gray_img, 100);
      printf("Wrote the gray image with a width of %dpx, a height of %dpx and %d channels\n", width, height, gray_channels);

      unsigned char *edge_img = (unsigned char*) malloc(img_size_gray);
 
      canny_edge_detect(gray_img, height, width, thresh_max, thresh_min,edge_img);
 
      stbi_write_jpg("sky_edge.jpeg", width, height, gray_channels, edge_img, 100);
      printf("Wrote the edge image with a width of %dpx, a height of %dpx and %d channels\n", width, height, gray_channels);

      stbi_image_free(img);
}